#include "hip/hip_runtime.h"
#include <iostream>
#include <time.h>

//chech if gpu is available
#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )
#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )
void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
        file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

//Allocates chunk to be reandred by thread block
__global__ void render(float *fb, int max_x, int max_y) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j*max_x*3 + i*3;
    fb[pixel_index + 0] = float(i) / max_x;
    fb[pixel_index + 1] = float(j) / max_y;
    fb[pixel_index + 2] = 0.2;
}

ing main(){
    //size of frame
    int width = 1200; int height = 600; //resolution of the image

    //number of threads allocated to render block (frame buffer)
    int threadsX = 8; int threadsY = 8; //8*8 64 threads per render block

    std::cerr << "Rendering... \n";
    

    int total_pixels = width * height;
    size_t chunk_size = 3*total_pixels*sizeof(double);

    //allocate memeory for chunk (frame buffer)
    double *chunk;
    checkCudaErrors(hipMallocManaged((void **)&chunk, chunk_size));

    clock_t t1, t2;
    t1 = clock();

    //Render our chunk (frame buffer)
    dim3 blocks(width/threadsX+1, height/threadsY+1);
    dim3 threads(tx, ty);
    render<<<blocks, threads>>>(chunk, width, height); //create threads

    checkCudaErrors(hipGetLastError()); //check for errors
    checkCudaErrors(hipDeviceSynchronize()); //syncronize threads

    t2 = clock();
    double timer_seconds = ((double)(t2 - t1))/ CLOCKS_PER_SEC;
    std::cerr << timer_seconds << "seconds \n";

    //Output chucks as immage
    for(int j = height-1; j >= 0; j--){
        for (int i = 0; i < nx; i++) {
            size_t pixel_index = j*3*nx + i*3;
            float r = fb[pixel_index + 0];
            float g = fb[pixel_index + 1];
            float b = fb[pixel_index + 2];
            int ir = int(255.99*r);
            int ig = int(255.99*g);
            int ib = int(255.99*b);
            std::cout << ir << " " << ig << " " << ib << "\n";
        }
    }

    checkCudaErrors(hipFree(chuck)); //kill threads
}

